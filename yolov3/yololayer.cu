#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include "yololayer.h"
#include <stdio.h>

using namespace nvinfer1;

YoloLayerPlugin::YoloLayerPlugin(int class_num, int yolo_grid, int input_dim, int cuda_block, float anchors[6]) {
    class_num_ = class_num;
    yolo_grid_ = yolo_grid;
    input_dim_ = input_dim;
    cuda_block_ = cuda_block;
    memcpy(anchors_, anchors, 6 * sizeof(float));
}

YoloLayerPlugin::~YoloLayerPlugin() {}

// create the plugin at runtime from a byte stream
YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
{
    char *d = (char*)(data), *a = d;

    class_num_ = *reinterpret_cast<int*>(d);
    d += sizeof(int);
    yolo_grid_ = *reinterpret_cast<int*>(d);
    d += sizeof(int);
    input_dim_ = *reinterpret_cast<int*>(d);
    d += sizeof(int);
    cuda_block_ = *reinterpret_cast<int*>(d);
    d += sizeof(int);
    memcpy(anchors_, d, 6 * sizeof(float));
    d += 6 * sizeof(float);

    if (d != a + length) {
        fprintf(stderr, "deserialize yololayer plugin failed! \n");
    }
}

void YoloLayerPlugin::serialize(void* buffer)
{
    char* d = static_cast<char*>(buffer), *a = d;

    *reinterpret_cast<int*>(d) = class_num_;
    d += sizeof(int);
    *reinterpret_cast<int*>(d) = yolo_grid_;
    d += sizeof(int);
    *reinterpret_cast<int*>(d) = input_dim_;
    d += sizeof(int);
    *reinterpret_cast<int*>(d) = cuda_block_;
    d += sizeof(int);
    memcpy(d, anchors_, 6 * sizeof(float));
    d += 6 * sizeof(float);

    if (d != a + getSerializationSize()) {
        fprintf(stderr, "serialize yololayer plugin failed! \n");
    }
}

size_t YoloLayerPlugin::getSerializationSize()
{  
    return sizeof(int) * 4 + sizeof(float) * 6;
}

int YoloLayerPlugin::initialize()
{ 
    return 0;
}

Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims) {
    return Dims3(3 * yolo_grid_ * yolo_grid_, 1, 7);
}

__device__ float Logist(float data){ return 1./(1. + exp(-data)); };

__global__ void CalDetection(const float *input, float *output, int noElements,
        int yolo_grid, const float anchors[6], int classes, int input_dim) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= noElements) return;

    int total_grid = yolo_grid * yolo_grid;
    int info_len_i = 5 + classes;
    int info_len_o = 7;
    int input_col = idx;
    int out_row = input_col;

    for (int k = 0; k < 3; ++k) {
        int class_id = 0;
        float max_prob = 0.0;
        for (int i = 5; i < info_len_i; ++i) {
            float p = Logist(input[input_col + k * info_len_i * total_grid + i * total_grid]);
            if (p > max_prob) {
                max_prob = p;
                class_id = i - 5;
            }
        }

        int row = idx / yolo_grid;
        int col = idx % yolo_grid;

        //Location
        output[out_row * info_len_o * 3 + info_len_o * k + 0] = (col + Logist(input[input_col + k * info_len_i * total_grid + 0 * total_grid])) * input_dim / yolo_grid;
        output[out_row * info_len_o * 3 + info_len_o * k + 1] = (row + Logist(input[input_col + k * info_len_i * total_grid + 1 * total_grid])) * input_dim / yolo_grid;
        output[out_row * info_len_o * 3 + info_len_o * k + 2] = exp(input[input_col + k * info_len_i * total_grid + 2 * total_grid]) * anchors[2*k];
        output[out_row * info_len_o * 3 + info_len_o * k + 3] = exp(input[input_col + k * info_len_i * total_grid + 3 * total_grid]) * anchors[2*k + 1];
        output[out_row * info_len_o * 3 + info_len_o * k + 4] =  Logist(input[input_col + k * info_len_i * total_grid + 4 * total_grid]);
        output[out_row * info_len_o * 3 + info_len_o * k + 5] =  class_id;
        output[out_row * info_len_o * 3 + info_len_o * k + 6] =  max_prob;
    }
}

void YoloLayerPlugin::forwardGpu(const float *const * inputs, float * output, hipStream_t stream, int batchSize) {
    void* anchors_gpu;
    hipMalloc(&anchors_gpu, 6 * sizeof(float));
    hipMemcpy(anchors_gpu, anchors_, 6 * sizeof(float), hipMemcpyHostToDevice);

    int block_size = cuda_block_;
    int grid_size = (yolo_grid_ * yolo_grid_ * batchSize + block_size - 1) / block_size;
    CalDetection<<<grid_size, block_size>>>
        (inputs[0], output, yolo_grid_ * yolo_grid_ * batchSize, yolo_grid_, (float *)anchors_gpu, class_num_, input_dim_);

    hipFree(anchors_gpu);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch caldetect kernel (error code %s)!\n", hipGetErrorString(err));
    }
}

int YoloLayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream) {
    //assert(batchSize == 1);
    //GPU
    //CUDA_CHECK(hipStreamSynchronize(stream));
    forwardGpu((const float *const *)inputs, (float *)outputs[0], stream, batchSize);
    return 0;
}
